#include <stdio.h>

int main()
{
	hipDeviceProp_t dev;
	int dev_cnt = 0;
	hipGetDeviceCount (&dev_cnt);
	
	for(int i = 0; i < dev_cnt; i++)
	{
		hipGetDeviceProperties(&dev, i);
		printf("Device %d:\n", i);
		printf("\tName: %s\n",dev.name);
		printf("\tCompute Capability %d.%d\n",dev.major, dev.minor);
		printf("\tMultiprocessor Count: %d\n", dev.multiProcessorCount);
		printf("\tTotal Global Memory (GB): %0.2f\n", 1.0*dev.totalGlobalMem/(1024*1024*1024));
		printf("\tTotal Constant Memory (kB): %d\n", dev.totalConstMem/1024);
		printf("\tShared Memory per Block (kB): %d\n",dev.sharedMemPerBlock/1024);
		printf("\tRegisters per Block: %d\n", dev.regsPerBlock);
		printf("\tMax Threads per Block: %d\n",dev.maxThreadsPerBlock);
		printf("\tMax Threads per Multiprocessor: %d\n", dev.maxThreadsPerMultiProcessor);
		printf("\tWarp Size: %d\n", dev.warpSize);
		printf("\tMax Thread Dim: %d,%d,%d\n", dev.maxThreadsDim[0], dev.maxThreadsDim[1], dev.maxThreadsDim[2]);
		printf("\tMax Grid Dim: %d,%d,%d\n", dev.maxGridSize[0],dev.maxGridSize[1], dev.maxGridSize[2]);
		printf("\tMemory Pitch: %d\n", dev.memPitch);
		printf("\tL2 Cache Size: %d\n", dev.l2CacheSize);
		printf("\tClock Rate (kHz): %d\n",dev.clockRate);
		printf("\tMemory Clock Rate (kHz): %d\n", dev.memoryClockRate);
		printf("\tMemory Bus Width (bits): %d\n", dev.memoryBusWidth);
	}
	return 0;
}
