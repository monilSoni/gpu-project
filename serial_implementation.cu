
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

#define DISPLAY_ON 1
#define DISPLAY_OFF 0

void setGrid(int *grid, int N) {
	// In future, this function will set
	// the grid to whatever we want it to be
	// maybe with a string argument

	int dummy_grid[10][10] = {{0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
							  {0, 0, 0, 1, 1, 0, 0, 0, 0, 0},
							  {0, 0, 0, 0, 1, 0, 0, 0, 0, 0},
							  {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
							  {0, 0, 0, 0, 0, 0, 0, 0, 0, 0},
							  {0, 0, 0, 1, 1, 0, 0, 0, 0, 0},
							  {0, 0, 1, 1, 0, 0, 0, 0, 0, 0},
							  {0, 0, 0, 0, 0, 1, 0, 0, 0, 0},
							  {0, 0, 0, 0, 1, 0, 0, 0, 0, 0},
							  {0, 0, 0, 0, 0, 0, 0, 0, 0, 0}};

	for (int i = 0; i < 10; i++) {
		for (int j = 0; j < 10; j++) {
			grid[i*N + j] = dummy_grid[i][j];
		}
	}
}

void display(int *arr, int N) {

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			arr[i*N + j] ? printf("* ") : printf(". ");
		}
		printf("\n");
	}
	printf("\n");
}

int setValue(int *grid, int x, int y, int N) {

	int aliveNeighbours = 0;
	for (int i = -1; i <= 1; i++) {
		for (int j = -1; j <= 1; j++) {
			aliveNeighbours += grid[((x+i)*N) + (y+j)];
		}
	}
	aliveNeighbours -= grid[x*N + y];

	return aliveNeighbours == 3 || (aliveNeighbours == 2 && grid[x*N + y]);
}

void nextGen(int *grid, int *newgrid, int N) { 

    for (int i = 1; i < N - 1; i++) {
		for (int j = 1; j < N - 1; j++) {
			newgrid[i*N + j] = setValue(grid, i, j, N);
		}
	}

	for (int i = 1; i < N-1; i++) {
		for (int j = 1; j < N-1; j++) {
			grid[i*N + j] = newgrid[i*N + j];
		}
	}
}

float simulateSerial(int *grid, int N, int disp_var) {

	// calls the function nextGen for the new grid
	// Updates the grid, sleeps and then displays.
	// It does this for SIMULATE_TIME times,
	// and returns the total runtime

	int *newgrid;
	newgrid = (int *)malloc(N * N * sizeof(int *));

	if(disp_var) {
		display(grid, N);
	}

	float elapsed_time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    nextGen(grid, newgrid, N);
    // usleep(100000);
    if(disp_var) {
        display(grid, N);
    }

	hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventElapsedTime(&elapsed_time, start, stop);

	free(newgrid);
	return elapsed_time;
}

int main() {

	int N = 16384;

	int *grid;
	grid = (int *)malloc(N * N * sizeof(int ));
	
	memset(grid, 0, N * N * sizeof(int));
	
	setGrid(grid, N);
	// display(grid, N);

	float elapsed_time = simulateSerial(grid, N, DISPLAY_OFF);
	// display(grid, N);

	printf("%dx%d  %0.4f ms\n", N, N, elapsed_time);

	free(grid);

	return 0;
}
